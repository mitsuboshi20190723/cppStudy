#include "hip/hip_runtime.h"
/*
 *  2023.11.21
 *  hfd.cu
 *  ver.0.5
 *  Kunihito Mitsuboshi
 *  license(Apache-2.0) at http://www.apache.org/licenses/LICENSE-2.0
 */


#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_fp16.h>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"


/*
__global__ void but_sum_gpu(double *v, double *sum)
{
	int l = threadIdx.x;

	for(long i=0; i<v.size(); i++)
	{
		*sum += v[i];
	}
//error	std::cout << "sum=" << *sum << std::endl; in __global__ function.
}

void sum(std::vector<half> v)
{
	for(long i=0; i<v.size(); i++)
	{
		v[0] += v[i+1];
	}
}
*/


int main(int argc, char **argv)
{
	int *gpu;
	long l, len(10000);
	
	half h; std::vector<half> vh(len,1);
	float f; std::vector<float> vf(len,1);
	double d; std::vector<double> vd(len,1);

	if(argc == 1) h=f=d=0.0;
	else
	{
		if(/*std::isdigit(argv[1])*/ 1 == 0) h=f=d=0.0;
		else
		{
			h = (half)std::stof(argv[1]);
			f = std::stof(argv[1]);
			d = std::stod(argv[1]);
		}
	}


	std::cout << "size of half   : " << sizeof(half) << std::endl;
TIMER_START
//	for(l=0; l<vh.size(); l++) h = h + vh[l];
TIMER_STOP
	std::cout << std::endl;

	std::cout << "size of float  : " << sizeof(float) << std::endl;
TIMER_START
	for(l=0; l<vf.size(); l++) f += vf[l];
TIMER_STOP
	std::cout << std::endl;

	std::cout << "size of double : " << sizeof(double) << std::endl;
TIMER_START
	for(l=0; l<vd.size(); l++) d += vd[l];
TIMER_STOP
	std::cout << std::endl;


	hipMalloc((void**)&gpu, len*sizeof(double));
/*
	hipMemcpy(gpu, vd, len*sizeof(double), hipMemcpyHostToDevice);

//error	but_sum_gpu<<<int numBlocks(1), dim3 threadsPerBlock(2, 2)>>>(gpu, ans); inisialize in <<< >>>>.
	dim3 BPG(1, 1); dim3 TPB(R, C); but_sum_gpu<<<BPG, TPB>>>(gpu, ans);

	hipMemcpy(vd, gpu, len*sizeof(double), hipMemcpyDeviceToHost);
*/
	hipFree(gpu);

	return 0;
}
