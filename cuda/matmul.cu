#include "hip/hip_runtime.h"
/*
 *  2023.11.19
 *  matmul.cu
 *  ver.0.7
 *  Kunihito Mitsuboshi
 *  license(Apache-2.0) at http://www.apache.org/licenses/LICENSE-2.0
 */


#include <iostream>
#include <random>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"

class matrix
{
public:
	int row; /* Gyou */
	int col; /* Retsu */
	float *mat;
};


__global__ void mat_mul_0(matrix a, matrix b, matrix c)
{
	int row = threadIdx.x;
	int col = threadIdx.y;

	float sum = (float)0;
	for(int i=0; i<a.col; i++) sum += a.mat[row*a.col+i] * b.mat[i*b.row+col];
	c.mat[row*c.col+col] = sum;

	printf("row=%d, col=%d : %f\n", row, col, sum);
}

#define INNER_PRODUCT c.mat[row*c.col+col]=0; for(int i=0; i<a.col; i++) c.mat[row*c.col+col] += a.mat[row*a.col+i] * b.mat[i*b.row+col];

__global__ void mat_mul_gpu(matrix a, matrix b, matrix c)
{
	int row = threadIdx.x;
	int col = threadIdx.y;

	INNER_PRODUCT
}

void mat_mul_cpu(matrix a, matrix b, matrix c)
{
	for(int col=0; col<a.col; col++)
	{
		for(int row=0; row<b.row; row++)
		{
			INNER_PRODUCT
		}
	}
}

void set_matrix(matrix m)
{
	int i, j;

	std::random_device rnd;
	std::uniform_real_distribution<> r1(0, 1);
	for(i=0; i<m.col; i++) for(j=0; j<m.row; j++) m.mat[i*m.col+j] = (float)r1(rnd);
}

void get_matrix(matrix m)
{
	int i, j;
	for(i=0; i<m.col; i++)
	{
		for(j=0; j<m.row; j++)
		{
			std::cout << *(m.mat + i*m.col + j) << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int chk_arg(int argc, char **argv, int *r, int *d, int *c, int *gpu, int *cpu, int *print)
{
	int count = 0, num = 0;

	for(int i=1; i<argc; i++)
	{
		if(argv[i][0] == '-')
		{
			switch( argv[i][1] )
			{
			case 'g':
				*cpu = 0;
				break;
			case 'c':
				*gpu = 0;
				break;
			case 'p':
				*print = 1;
				break;
			default:
				std::cout << "-" << argv[i][1] << " : unknown option." << std::endl;
				exit(0);
			}
		}
		else
		{
			num = atoi(argv[i]);
			if(num > 0)
			{ 
				switch( count )
				{
				case 0:
					*r = atoi(argv[i]);
					break;
				case 1:
					*d = atoi(argv[i]);
					break;
				case 2:
					*c = atoi(argv[i]);
					break;
				}
				count++;
			}
			else
			{
				std::cout << "-" << argv[i] << " is but number." << std::endl;
				exit(0);
			}
		}
	}
	return count;
}

int main(int argc, char **argv)
{
	matrix a, b, c; /* for CPU */
	matrix d_a, d_b, d_c; /* for GPU */
	int R(1), D(1), C(1), GPU(1), CPU(1), PRINT(0);

	chk_arg(argc, argv, &R, &D, &C, &GPU, &CPU, &PRINT);


	a.row = c.row = R;
	a.col = b.row = D;
	b.col = c.col = C;

	d_a.row = a.row; d_a.col = a.col;
	d_b.row = b.row; d_b.col = b.col;
	d_c.row = c.row; d_c.col = c.col;

	hipMalloc((void**)&d_a.mat, d_a.row*d_a.col*sizeof(float));
	hipMalloc((void**)&d_b.mat, d_b.row*d_b.col*sizeof(float));
	hipMalloc((void**)&d_c.mat, d_c.row*d_c.col*sizeof(float));
	a.mat = new float[a.row*a.col];
	b.mat = new float[b.row*b.col];
	c.mat = new float[c.row*c.col];

	std::cout << "Matrix a (" << a.row << " rows, " << a.col << " columns)" << std::endl;
	set_matrix(a); if(PRINT) get_matrix(a);
	std::cout << "Matrix b (" << b.row << " rows, " << b.col << " columns)" << std::endl;
	set_matrix(b); if(PRINT) get_matrix(b);

	if(GPU)
	{
		std::cout << "\n----- GPU -----" << std::endl;

TIMER_START
		hipMemcpy(d_a.mat, a.mat, a.row*a.col*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_b.mat, b.mat, b.row*b.col*sizeof(float), hipMemcpyHostToDevice);
		dim3 BPG(1, 1); dim3 TPB(R, C); mat_mul_gpu<<<BPG, TPB>>>(d_a, d_b, d_c);
		hipMemcpy(c.mat, d_c.mat, d_c.row*d_c.col*sizeof(float), hipMemcpyDeviceToHost);
TIMER_STOP

		std::cout << "Matrix c=a*b (" << c.row << "rows, " << c.col << "columns)" << std::endl;
		if(PRINT) get_matrix(c);
	}


	if(CPU)
	{
		std::cout << "\n----- CPU -----" << std::endl;
TIMER_START
		mat_mul_cpu(a, b, c);
TIMER_STOP
		std::cout << "Matrix c=a*b (" << c.row << "rows, " << c.col << "columns)" << std::endl;
		if(PRINT) get_matrix(c);
	}


	hipFree(d_a.mat);
	hipFree(d_b.mat);
	hipFree(d_c.mat);
	delete[] a.mat;
	delete[] b.mat;
	delete[] c.mat;

	return 0;
}
