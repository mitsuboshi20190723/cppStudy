#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void MatAdd(float *A, float *B, float *C)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
//	C[i][j] = A[i][j] + B[i][j];
	*(C+i*j) = *(A+i*j) + *(B+i*j);
}

int main(int argc, char *argv[])
{
	float *A, *B, *C;
	int N = 2;
	int i, j;

	hipMalloc((void**)&A, N*N*sizeof(float));
	hipMalloc((void**)&B, N*N*sizeof(float));
	hipMalloc((void**)&C, N*N*sizeof(float));
	float *a = (float *)malloc(N*N*sizeof(float));
	float *b = (float *)malloc(N*N*sizeof(float));
	float *c = (float *)malloc(N*N*sizeof(float));

	for(i=0; i<N; i++){ for(j=0; j<N; j++){ *(a+i*N+j) = (float)random(); *(b+i*N+j) = (float)random(); }}
	hipMemcpy(A, a, N*N*sizeof(*A), hipMemcpyHostToDevice);
	hipMemcpy(B, b, N*N*sizeof(*B), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);

	hipMemcpy(c, C, N*N*sizeof(*c), hipMemcpyDeviceToHost);
	for(i=0; i<N; i++){ for(j=0; j<N; j++){ printf("i=%4d, j=%4d :%14.1f +%14.1f =%14.1f :%14.1f\n", i, j, *(a+i*N+j), *(b+i*N+j), *(c+i*N+j), *(a+i*N+j)+*(b+i*N+j)); }}

	hipFree(A);
	hipFree(B);
	hipFree(C);
	free(a);
	free(b);
	free(c);

	return 0;
}
